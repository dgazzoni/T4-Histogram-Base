
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255

typedef struct {
  unsigned char red, green, blue;
} PPMPixel;

typedef struct {
  int x, y;
  PPMPixel *data;
} PPMImage;

static PPMImage *readPPM(const char *filename) {
  char buff[16];
  PPMImage *img;
  FILE *fp;
  int c, rgb_comp_color;
  fp = fopen(filename, "rb");
  if (!fp) {
    fprintf(stderr, "Unable to open file '%s'\n", filename);
    exit(1);
  }

  if (!fgets(buff, sizeof(buff), fp)) {
    perror(filename);
    exit(1);
  }

  if (buff[0] != 'P' || buff[1] != '6') {
    fprintf(stderr, "Invalid image format (must be 'P6')\n");
    exit(1);
  }

  img = (PPMImage *)malloc(sizeof(PPMImage));
  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  c = getc(fp);
  while (c == '#') {
    while (getc(fp) != '\n')
      ;
    c = getc(fp);
  }

  ungetc(c, fp);
  if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
    fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
    exit(1);
  }

  if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
    fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
    exit(1);
  }

  if (rgb_comp_color != RGB_COMPONENT_COLOR) {
    fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
    exit(1);
  }

  while (fgetc(fp) != '\n')
    ;
  img->data = (PPMPixel *)malloc(img->x * img->y * sizeof(PPMPixel));

  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
    fprintf(stderr, "Error loading image '%s'\n", filename);
    exit(1);
  }

  fclose(fp);
  return img;
}

__global__ void Histogram(PPMImage *image, float *h) {
  printf("TODO: Implement this kernel!\n");
}

int main(int argc, char *argv[]) {

  double t_start, t_end;
  int i;
  char filename[255];

  if (argc < 2) {
    fprintf(stderr, "Error: missing path to input file\n");
    return 1;
  }

  PPMImage *image = readPPM(argv[1]);

  float *h = (float *)malloc(sizeof(float) * 64);

  // Inicializar h
  for (i = 0; i < 64; i++)
    h[i] = 0.0;

  t_start = omp_get_wtime();
  Histogram<<<1, 1>>>(image, h);
  t_end = omp_get_wtime();

  for (i = 0; i < 64; i++) {
    printf("%0.3f ", h[i]);
  }
  printf("\n");
  fprintf(stderr, "%lf\n", t_end - t_start);

  free(h);
}
